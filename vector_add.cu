
#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>
using namespace std;

__global__ void vecAdd(int *A, int *B, int *C, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size){
        C[tid] = A[tid] + B[tid];
    }
}

int main(){

    int size = 5;
    vector<int> A = {1,2,3,4,5};
    vector<int> B = {2,3,4,5,6};
    vector<int> C;
    C.resize(size);

    int allocSize = size*sizeof(int);
    int *X, *Y, *Z;

    hipMalloc(&X, allocSize);
    hipMalloc(&Y, allocSize);
    hipMalloc(&Z, allocSize);

    hipMemcpy(X, A.data(), allocSize, hipMemcpyHostToDevice);
    hipMemcpy(Y, B.data(), allocSize, hipMemcpyHostToDevice);

    vecAdd<<<1, 255>>>(X, Y, Z, size);

    hipMemcpy(C.data(), Z, allocSize, hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++){
        cout<<C[i]<<" ";
    }

    return 0;
}