
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
using namespace std;

#define N 3  // Matrix size N x N

// CUDA kernel for matrix multiplication
__global__ void matMul(int* A, int* B, int* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Column index

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; ++k)
            sum += A[row * n + k] * B[k * n + col];

        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N;
    int bytes = size * sizeof(int);

    // Host matrices
    vector<int> h_A = {
        1, 2, 3,
        4, 5, 6,
        7, 8, 9
    };
    vector<int> h_B = {
        9, 8, 7,
        6, 5, 4,
        3, 2, 1
    };
    vector<int> h_C(size);  // Result matrix

    // Device matrices
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data to device
    hipMemcpy(d_A, h_A.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), bytes, hipMemcpyHostToDevice);

    // Kernel launch: one thread per element
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    matMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back
    hipMemcpy(h_C.data(), d_C, bytes, hipMemcpyDeviceToHost);

    // Print result
    cout << "Result matrix C:" << endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j)
            cout << h_C[i * N + j] << "\t";
        cout << endl;
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
